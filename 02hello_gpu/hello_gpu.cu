#include "hip/hip_runtime.h"
/**
   @file hello_gpu.cu
  */
#include <assert.h>
#include <stdio.h>

/*

  warning : the following code does not check any error returned from
  API calls or kernel launches to keep the textual complexity of the
  program low.  This is a highly discouraged practice.  When you don't
  check errors from kernel launches, your program keeps running and 
  you notice it by wrong results.  

  This code is just to illustrate the concepts you need to master when
  writing CUDA programs.


 */

__global__ void cuda_thread_fun(int n) {
  int i        = blockDim.x * blockIdx.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;
  if (i < n) {
    printf("hello I am CUDA thread %d out of %d\n", i, nthreads);
  }
}

int main(int argc, char ** argv) {
  int n               = (argc > 1 ? atoi(argv[1]) : 100);
  int thread_block_sz = (argc > 2 ? atoi(argv[2]) : 64);
  int n_thread_blocks = (n + thread_block_sz - 1) / thread_block_sz;

  cuda_thread_fun<<<n_thread_blocks,thread_block_sz>>>(n);
  hipError_t e = hipGetLastError();
  if (e) {
    printf("NG: %s\n", hipGetErrorString(e)); return 1;
  } else {
    printf("OK\n");
  }
  e = hipDeviceSynchronize();
  if (e) {
    printf("NG: %s\n", hipGetErrorString(e)); return 1;
  } else {
    printf("OK\n");
  }
  return 0;
}
